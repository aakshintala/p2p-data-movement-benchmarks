#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <fstream>
#include <sstream>
#include <memory>
#include <string>
#include <cstring>
#include <cstdlib>
#include <cstdio>
#include <vector>
#include <deque>
#include <atomic>
#include <mutex>
#include <condition_variable>
#include <unordered_map>
#include <thread>
#include <cmath>
#include <sys/stat.h>
#include <sys/time.h>
#include <sys/wait.h>
#include <sys/types.h>
#include <functional>
#include <fcntl.h>
#include <unistd.h>
#include <signal.h>

#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime_api.h>
#include "nvToolsExt.h"

#include <assert.h>

// Gotta createt the logger before including FFmpegStreamer/Demuxer
#include "Logger.h"
simplelogger::Logger *logger = simplelogger::LoggerFactory::CreateConsoleLogger();

#include <cudaUtils.h>
#include <Timer.h>

using namespace std;

enum copyMode
{
	memcpyThroughHostPinned,
	memcpyThroughHostUnpinned,
	memcpyP2P,
	copyKernelNVLINK,
	copyKernelUVM
};

__global__ void delay(volatile int *flag, unsigned long long timeout_clocks = 10000000)
{
	// Wait until the application notifies us that it has completed queuing up the
	// experiment, or timeout and exit, allowing the application to make progress
	long long int start_clock, sample_clock;
	start_clock = clock64();

	while (!*flag) {
		sample_clock = clock64();

		if (sample_clock - start_clock > timeout_clocks) {
			break;
		}
	}
}

// This kernel is for demonstration purposes only, not a performant kernel for p2p transfers.
__global__ void copyp2p(int4* __restrict__  dest, int4 const* __restrict__ src, size_t num_elems)
{
	size_t globalId = blockIdx.x * blockDim.x + threadIdx.x;
	size_t gridSize = blockDim.x * gridDim.x;

	#pragma unroll(5)
	for (size_t i=globalId; i < num_elems; i+= gridSize)
	{
		dest[i] = src[i];
	}
}


void copyKernel(int *dest, int destDevice, int *src, int srcDevice, int bufferSize, int repeat,
				hipStream_t streamToRun)
{
	int blockSize = 0;
	int numBlocks = 0;

	CUDA_ASSERT(hipOccupancyMaxPotentialBlockSize(&numBlocks, &blockSize, copyp2p));

	for (int r = 0; r < repeat; r++)
		copyp2p<<<numBlocks, blockSize, 0, streamToRun>>>((int4*)dest, (int4*)src,
															bufferSize/(4*sizeof(int)));
}

void measureBandwidthAndUtilization(int numGPUs, int numElems, int objectSize, copyMode mode)
{
	int repeat = 5;
	bool p2p = false;
	size_t bufferSize = numElems * objectSize;
	volatile int *flag = NULL;
	vector<int *> buffers(numGPUs);
	vector<int *> buffersHost(numGPUs);
	vector<int *> buffersD2D(numGPUs); // buffer for D2D, that is, intra-GPU copy
	vector<hipEvent_t> start(numGPUs);
	vector<hipEvent_t> stop(numGPUs);
	vector<hipStream_t> stream(numGPUs);

	switch(mode) {
		case memcpyThroughHostPinned:
			for (int i = 0; i < numGPUs; i++)
				CUDA_ASSERT(hipHostMalloc(&buffersHost[i], bufferSize));
			break;
		case memcpyThroughHostUnpinned:
			for (int i = 0; i < numGPUs; i++)
				buffersHost[i] = new int[bufferSize];
			break
		case memcpyP2P:
		case copyKernelNVLINK:
			p2p = true;
			break
		case copyKernelUVM:
			p2p = false;
			break
	}

	CUDA_ASSERT(hipHostAlloc((void **)&flag, sizeof(*flag), hipHostMallocPortable));

	for (int d = 0; d < numGPUs; d++) {
		hipSetDevice(d);
		hipStreamCreateWithFlags(&stream[d], hipStreamNonBlocking);
		CUDA_ASSERT(hipMalloc(&buffers[d], bufferSize));
		CUDA_ASSERT(hipMalloc(&buffersD2D[d], bufferSize));
		CUDA_ASSERT(hipEventCreate(&start[d]));
		CUDA_ASSERT(hipEventCreate(&stop[d]));
	}

	vector<double> bandwidthMatrix(numGPUs * numGPUs);

	for (int i = 0; i < numGPUs; i++) {
		hipSetDevice(i);

		for (int j = 0; j < numGPUs; j++) {
			if (i == j) {
				bandwidthMatrix[i * numGPUs + j] = -1.0;
				continue;
			}
			int access = 0;
			if (p2p) {
				hipDeviceCanAccessPeer(&access, i, j);
				if (access) {
					CUDA_ASSERT(hipDeviceEnablePeerAccess(j, 0));
					CUDA_ASSERT(hipSetDevice(j));
					CUDA_ASSERT(hipDeviceEnablePeerAccess(i, 0));
					CUDA_ASSERT(hipSetDevice(i));
				}
			}

			hipStreamSynchronize(stream[i]);
			cudaCheckError();

			// Block the stream until all the work is queued up
			// DANGER! - hipMemcpy*Async may infinitely block waiting for
			// room to push the operation, so keep the number of repetitions
			// relatively low.  Higher repetitions will cause the delay kernel
			// to timeout and lead to unstable results.
			*flag = 0;
			CUDA_ASSERT(delay<<< 1, 1, 0, stream[i]>>>(flag));


			CUDA_ASSERT(hipEventRecord(start[i], stream[i]));
			for (int r = 0; r < repeat; r++) {
				switch(mode) {
					case memcpyThroughHostPinned:
					case memcpyThroughHostUnpinned:
						CUDA_ASSERT(hipMemcpyAsync((void *)buffersHost[i], (const void*)buffers[i], bufferSize, hipMemcpyDeviceToHost, stream[i]));
						CUDA_ASSERT(hipMemcpyAsync((void *)buffers[j], (const void*)buffersHost[i], bufferSize, hipMemcpyHostToDevice, stream[i]));
						break
					case memcpyP2P:
						CUDA_ASSERT(hipMemcpyPeerAsync((void *)buffers[j], j, (const void*) buffers[i], i, bufferSize, stream[i]))
						break
					case copyKernelNVLINK:
					case copyKernelUVM:
						copyKernel(buffers[i], i, buffers[j], j, bufferSize, repeat, stream[i]);
						break
				}
			}
			CUDA_ASSERT(hipEventRecord(stop[i], stream[i]));

			// Release the queued events
			*flag = 1;
			CUDA_ASSERT(hipStreamSynchronize(stream[i]));

			float time_ms;
			hipEventElapsedTime(&time_ms, start[i], stop[i]);
			double time_s = time_ms / 1e3;

			double gb = numElems * objectSize * repeat / (double)1e9;
			bandwidthMatrix[i * numGPUs + j] = gb / time_s;

			if (p2p && access) {
				CUDA_ASSERT(hipDeviceDisablePeerAccess(j));
				CUDA_ASSERT(hipSetDevice(j));
				CUDA_ASSERT(hipDeviceDisablePeerAccess(i));
				CUDA_ASSERT(hipSetDevice(i));
			}
		}
	}

	sstream header;
	for (int j = 0; j < numGPUs; j++) {
		header <<std::setw(6) <<j;
	}
	LOG(INFO) << header;

	sstream row;
	for (int i = 0; i < numGPUs; i++) {
		row <<std::setw(6) <<i;

		for (int j = 0; j < numGPUs; j++) {
			if (i == j)
				row <<std::setw(6) <<"-";
			row <<std::setw(6) <<std::setprecision(2) << bandwidthMatrix[i * numGPUs + j];
		}
		LOG(INFO) << row;
	}

	for (int d = 0; d < numGPUs; d++) {
		hipSetDevice(d);
		CUDA_ASSERT(hipFree(buffers[d]));
		CUDA_ASSERT(hipFree(buffersD2D[d]));
		CUDA_ASSERT(hipEventDestroy(start[d]));
		CUDA_ASSERT(hipEventDestroy(stop[d]));
		CUDA_ASSERT(hipStreamDestroy(stream[d]));
	}

	switch(mode) {
		case memcpyThroughHostPinned:
			for (int i = 0; i < numGPUs; i++)
				CUDA_ASSERT(hipFree(buffersHost[i]));
			break;
		case memcpyThroughHostUnpinned:
			for (int i = 0; i < numGPUs; i++)
				delete [] buffersHost[i];
			break
	}

	CUDA_ASSERT(hipHostFree((void *)flag));
}

void checkP2Paccess(int numGPUs)
{
	for (int i = 0; i < numGPUs; i++) {
		CUDA_ASSERT(hipSetDevice(i));

		for (int j = 0; j < numGPUs; j++) {
			int access;
			if (i != j) {
				CUDA_ASSERT(hipDeviceCanAccessPeer(&access, i, j));
				LOG(INFO)<< "Device "<<i  <<access? "CAN" : "CANNOT" <<" access peer device " <<j;
			}
		}
	}
	LOG(INFO) << "***NOTE: When a device doesn't have P2P access, it falls back to cudaMemCpyAsync through the host, in which case, you'll observe a loss in bandwidth (GB/s) and higher latency (us).***";
}

int main(int argc, char **argv)
{
	int numGPUs;
	int queueDepth = 1;
	int objectSize = sizeof(int);

	CUDA_ASSERT(hipGetDeviceCount(&numGPUs));


	//process command line args
	for (int i = 1; i < argc; i++) {
		if(0==strcmp(argv[i], "-h")){
			LOG(INFO)<< "Usage:" << argv[0] " [OPTION]..." << std::endl
					<< "Options:" <<std::endl
					<< "-h\tDisplay this Help menu" <<std::endl
					<< "-q\tQueue depth" <<std::endl
					<< "-s\tobject Size";
			return 0;
		} else if (0==strcmp(argv[i], "-q")) {
			queueDepth = atoi(argv[i+1]);
		} else if (0==strcmp(argv[i], "-q")) {
			objectSize = atoi(argv[i+1]);
		}
	}

	LOG(INFO) << "GPU to GPU Bandwidth & Latency Test\n";

	//output devices
	for (int i = 0; i < numGPUs; i++) {
		hipDeviceProp_t prop;
		CUDA_ASSERT(hipGetDeviceProperties(&prop, i));
		LOG(INFO) << "Device:" << i <<" " <<prop.name <<" pciBusID:" <<std::hex <<prop.pciBusID <<"pciDeviceID:" <<prop.pciDeviceID <<"pciDomainID:" << prop.pciDomainID;
	}

	checkP2Paccess(numGPUs);
	measureBandwidthAndUtilization(numGPUs, queueDepth, objectSize, memcpyThroughHostPinned);
	measureBandwidthAndUtilization(numGPUs, queueDepth, objectSize, memcpyThroughHostUnpinned);
	measureBandwidthAndUtilization(numGPUs, queueDepth, objectSize, memcpyP2P);
	measureBandwidthAndUtilization(numGPUs, queueDepth, objectSize, copyKernelNVLINK);
	measureBandwidthAndUtilization(numGPUs, queueDepth, objectSize, copyKernelUVM);
	exit(EXIT_SUCCESS);
}
