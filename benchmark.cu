#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <fstream>
#include <sstream>
#include <memory>
#include <string>
#include <cstring>
#include <cstdlib>
#include <cstdio>
#include <vector>
#include <deque>
#include <atomic>
#include <mutex>
#include <condition_variable>
#include <unordered_map>
#include <thread>
#include <cmath>
#include <sys/stat.h>
#include <sys/time.h>
#include <sys/wait.h>
#include <sys/types.h>
#include <functional>
#include <fcntl.h>
#include <unistd.h>
#include <signal.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime_api.h>
#include "nvToolsExt.h"

#include <assert.h>

// Gotta createt the logger before including FFmpegStreamer/Demuxer
#include "Logger.h"
simplelogger::Logger *logger = simplelogger::LoggerFactory::CreateConsoleLogger();

#include "cudaUtils.h"
#include "Timer.h"

using namespace std;

enum copyMode
{
	memcpyThroughHostPinned,
	memcpyThroughHostUnpinned,
	memcpyP2P,
	copyKernelNVLINK,
	copyKernelUVM
};

__global__ void delay(volatile int *flag, unsigned long long timeout_clocks = 10000000)
{
	// Wait until the application notifies us that it has completed queuing up the
	// experiment, or timeout and exit, allowing the application to make progress
	long long int start_clock, sample_clock;
	start_clock = clock64();

	while (!*flag) {
		sample_clock = clock64();

		if (sample_clock - start_clock > timeout_clocks) {
			break;
		}
	}
}

// This kernel is for demonstration purposes only, not a performant kernel for p2p transfers.
__global__ void incKernel(int*  buffer, size_t num_elems)
{
	size_t globalId = blockIdx.x * blockDim.x + threadIdx.x;
	size_t gridSize = blockDim.x * gridDim.x;

	#pragma unroll(5)
	for (size_t i=globalId; i < num_elems; i+= gridSize)
	{
		buffer[i] += 1;
	}
}

void incBuffer(int *buffer, int bufferSize, hipStream_t streamToRun)
{
	int blockSize = 0;
	int numBlocks = 0;

	CUDA_ASSERT(hipOccupancyMaxPotentialBlockSize(&numBlocks, &blockSize, incKernel));

	incKernel<<<numBlocks, blockSize, 0, streamToRun>>>((int*)buffer,bufferSize/(sizeof(int)));
}

// This kernel is for demonstration purposes only, not a performant kernel for p2p transfers.
__global__ void copyp2p(int4* __restrict__  dest, int4 const* __restrict__ src, size_t num_elems)
{
	size_t globalId = blockIdx.x * blockDim.x + threadIdx.x;
	size_t gridSize = blockDim.x * gridDim.x;

	#pragma unroll(5)
	for (size_t i=globalId; i < num_elems; i+= gridSize)
	{
		dest[i] = src[i];
	}
}


void copyKernel(int *dest, int destDevice, int *src, int srcDevice, int bufferSize, int repeat,
				hipStream_t streamToRun)
{
	int blockSize = 0;
	int numBlocks = 0;

	CUDA_ASSERT(hipOccupancyMaxPotentialBlockSize(&numBlocks, &blockSize, copyp2p));

	for (int r = 0; r < repeat; r++)
		copyp2p<<<numBlocks, blockSize, 0, streamToRun>>>((int4*)dest, (int4*)src,
															bufferSize/(4*sizeof(int)));
}

void measureBandwidthAndUtilization(int numGPUs, int numElems, int objectSize, copyMode mode)
{
	int repeat = 5;
	bool p2p = false;
	uint64_t bufferSize = numElems * objectSize;
	volatile int *flag = NULL;
	vector<int *> buffers(numGPUs);
	vector<int *> buffersHost(numGPUs);
	vector<int *> buffersD2D(numGPUs); // buffer for D2D, that is, intra-GPU copy
	vector<hipEvent_t> start(numGPUs);
	vector<hipEvent_t> stop(numGPUs);
	vector<hipStream_t> stream(numGPUs);

	switch(mode) {
		case memcpyThroughHostPinned:
			for (int i = 0; i < numGPUs; i++)
				CUDA_ASSERT(hipHostMalloc(&buffersHost[i], bufferSize));
			break;
		case memcpyThroughHostUnpinned:
			for (int i = 0; i < numGPUs; i++)
				buffersHost[i] = new int[bufferSize];
			break;
		case memcpyP2P:
		case copyKernelNVLINK:
			p2p = true;
			break;
		case copyKernelUVM:
			p2p = false;
			for (int i = 0; i < numGPUs; i++)
				CUDA_ASSERT(hipMallocManaged(&buffersHost[i], bufferSize));
			break;
	}

	CUDA_ASSERT(hipHostAlloc((void **)&flag, sizeof(*flag), hipHostMallocPortable));

	for (int d = 0; d < numGPUs; d++) {
		hipSetDevice(d);
		hipStreamCreateWithFlags(&stream[d], hipStreamNonBlocking);
		CUDA_ASSERT(hipMalloc(&buffers[d], bufferSize));
		CUDA_ASSERT(hipMalloc(&buffersD2D[d], bufferSize));
		CUDA_ASSERT(hipEventCreate(&start[d]));
		CUDA_ASSERT(hipEventCreate(&stop[d]));

		// Make the buffers GPU-resident if they are UVM-managed.
		if (mode == copyKernelUVM)
			incBuffer(buffersHost[d], bufferSize, stream[d]);
	}

	vector<double> bandwidthMatrix(numGPUs * numGPUs);

	for (int i = 0; i < numGPUs; i++) {
		hipSetDevice(i);

		for (int j = 0; j < numGPUs; j++) {
			if (i == j) {
				bandwidthMatrix[i * numGPUs + j] = -1.0;
				continue;
			}
			int access = 0;
			if (p2p) {
				hipDeviceCanAccessPeer(&access, i, j);
				if (access) {
					CUDA_ASSERT(hipDeviceEnablePeerAccess(j, 0));
					CUDA_ASSERT(hipSetDevice(j));
					CUDA_ASSERT(hipDeviceEnablePeerAccess(i, 0));
					CUDA_ASSERT(hipSetDevice(i));
				}
			}

			CUDA_ASSERT(hipStreamSynchronize(stream[i]));

			// Block the stream until all the work is queued up
			// DANGER! - hipMemcpy*Async may infinitely block waiting for
			// room to push the operation, so keep the number of repetitions
			// relatively low.  Higher repetitions will cause the delay kernel
			// to timeout and lead to unstable results.
			*flag = 0;
			delay<<< 1, 1, 0, stream[i]>>>(flag);

			float time_ms;

			CUDA_ASSERT(hipEventRecord(start[i], stream[i]));
			switch(mode) {
				case memcpyThroughHostPinned:
				case memcpyThroughHostUnpinned:
					for (int r = 0; r < repeat; r++) {
						CUDA_ASSERT(hipMemcpyAsync((void *)buffersHost[i], (const void*)buffers[i], bufferSize, hipMemcpyDeviceToHost, stream[i]));
						CUDA_ASSERT(hipMemcpyAsync((void *)buffers[j], (const void*)buffersHost[i], bufferSize, hipMemcpyHostToDevice, stream[i]));
					}
					break;
				case memcpyP2P:
					for (int r = 0; r < repeat; r++)
						CUDA_ASSERT(hipMemcpyPeerAsync((void *)buffers[j], j, (const void*) buffers[i], i, bufferSize, stream[i]));
					break;
				case copyKernelNVLINK:
					for (int r = 0; r < repeat; r++)
						copyKernel(buffers[i], i, buffers[j], j, bufferSize, repeat, stream[i]);
					break;
				case copyKernelUVM:
					// Copy from and to UVM managed buffers
					incBuffer(buffersHost[j], bufferSize, stream[i]);
					CUDA_ASSERT(hipEventRecord(start[i], stream[i]));
					copyKernel(buffers[i], i, buffersHost[j], j, bufferSize, repeat, stream[i]);
					break;
			}
			CUDA_ASSERT(hipEventRecord(stop[i], stream[i]));

			// Release the queued events
			*flag = 1;
			CUDA_ASSERT(hipStreamSynchronize(stream[i]));

			hipEventElapsedTime(&time_ms, start[i], stop[i]);
			double time_s = time_ms / 1e3;

			double gb = 0.0;
			if (copyKernelUVM==mode)
				gb = bufferSize / (double)1e9;
			else
				gb = bufferSize * repeat / (double)1e9;
			bandwidthMatrix[i * numGPUs + j] = gb / time_s;

			if (p2p && access) {
				CUDA_ASSERT(hipDeviceDisablePeerAccess(j));
				CUDA_ASSERT(hipSetDevice(j));
				CUDA_ASSERT(hipDeviceDisablePeerAccess(i));
				CUDA_ASSERT(hipSetDevice(i));
			}
		}
	}

	LOG(INFO) <<std::setw(6) <<" ";
	for (int j = 0; j < numGPUs; j++) {
		LOG(INFO) <<std::setw(6) <<j;
	}
	LOG(INFO) << std::endl;

	for (int i = 0; i < numGPUs; i++) {
		LOG(INFO) <<std::setw(6) <<i;

		for (int j = 0; j < numGPUs; j++) {
			if (i == j)
				LOG(INFO) <<std::setw(6) <<"-";
			else
				LOG(INFO) <<std::setw(6) <<std::setprecision(2) << bandwidthMatrix[i * numGPUs + j];
		}
		LOG(INFO) << std::endl;
	}
	LOG(INFO) << std::endl;

	for (int d = 0; d < numGPUs; d++) {
		hipSetDevice(d);
		CUDA_ASSERT(hipFree(buffers[d]));
		CUDA_ASSERT(hipFree(buffersD2D[d]));
		CUDA_ASSERT(hipEventDestroy(start[d]));
		CUDA_ASSERT(hipEventDestroy(stop[d]));
		CUDA_ASSERT(hipStreamDestroy(stream[d]));
	}

	switch(mode) {
		case memcpyThroughHostPinned:
			for (int i = 0; i < numGPUs; i++)
				CUDA_ASSERT(hipHostFree(buffersHost[i]));
			break;
		case memcpyThroughHostUnpinned:
			for (int i = 0; i < numGPUs; i++)
				delete [] buffersHost[i];
			break;
		case copyKernelUVM:
			for (int i = 0; i < numGPUs; i++)
				CUDA_ASSERT(hipFree(buffersHost[i]));
			break;
		default:
			break;
	}

	CUDA_ASSERT(hipHostFree((void *)flag));
}

void checkP2Paccess(int numGPUs)
{
	for (int i = 0; i < numGPUs; i++) {
		CUDA_ASSERT(hipSetDevice(i));

		for (int j = 0; j < numGPUs; j++) {
			int access = 0;
			if (i != j) {
				CUDA_ASSERT(hipDeviceCanAccessPeer(&access, i, j));
				LOG(INFO)<< "Device "<<i  <<(access? " CAN" : " CANNOT") <<" access peer device " <<j <<std::endl;
			}
		}
	}
	LOG(INFO) << "***NOTE: When a device doesn't have P2P access, it falls back to cudaMemCpyAsync through the host, in which case, you'll observe a loss in bandwidth (GB/s) and higher latency (us).***" << std::endl;
}

int main(int argc, char **argv)
{
	int numGPUs = 0;
	int queueDepth = 1024*1024;
	int objectSize = sizeof(int);

	CUDA_ASSERT(hipGetDeviceCount(&numGPUs));
	assert(numGPUs != 0);
	LOG(INFO) << "Moving " << queueDepth*objectSize << "bytes of data" << std::endl;
	//process command line args
	for (int i = 1; i < argc; i++) {
		if (0==strcmp(argv[i], "-h")) {
			LOG(ERROR) << "Usage:" << argv[0] <<" [OPTION]..." <<std::endl << "Options:" <<std::endl << "-h\tDisplay this Help menu" <<std::endl <<"-q\tQueue depth" <<std::endl << "-s\tobject Size" << std::endl;
			return 0;
		} else if (0==strcmp(argv[i], "-q")) {
			queueDepth = atoi(argv[i+1]);
		} else if (0==strcmp(argv[i], "-s")) {
			objectSize = atoi(argv[i+1]);
		}
	}

	LOG(INFO) << "GPU to GPU Bandwidth & Latency Test\n";

	//output devices
	for (int i = 0; i < numGPUs; i++) {
		hipDeviceProp_t prop;
		CUDA_ASSERT(hipGetDeviceProperties(&prop, i));
		LOG(INFO) << "Device:" << i <<" " <<prop.name <<" pciBusID:" <<std::hex <<prop.pciBusID <<" pciDeviceID:" <<std::hex <<prop.pciDeviceID <<" pciDomainID:" <<std::hex << prop.pciDomainID <<std::endl <<std::dec;
	}

	checkP2Paccess(numGPUs);
	LOG(INFO) <<"\nmemcpyThroughHostPinned\n";
	measureBandwidthAndUtilization(numGPUs, queueDepth, objectSize, memcpyThroughHostPinned);
	LOG(INFO) <<"\nmemcpyThroughHostUnpinned\n";
	measureBandwidthAndUtilization(numGPUs, queueDepth, objectSize, memcpyThroughHostUnpinned);
	LOG(INFO) <<"\nmemcpyP2P\n";
	measureBandwidthAndUtilization(numGPUs, queueDepth, objectSize, memcpyP2P);
	LOG(INFO) <<"\ncopyKernelNVLINK\n";
	measureBandwidthAndUtilization(numGPUs, queueDepth, objectSize, copyKernelNVLINK);
	LOG(INFO) <<"\ncopyKernelUVM\n";
	measureBandwidthAndUtilization(numGPUs, queueDepth, objectSize, copyKernelUVM);
	exit(EXIT_SUCCESS);
}
